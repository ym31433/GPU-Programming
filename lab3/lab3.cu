#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	const int yb = oy+yt, xb = ox+xt;
	const int curb = wb*yb+xb;
	float temp[3];
    if(0 <= yb and yb < hb and 0 <= xb and xb < wb) {
        temp[0] = output[curb*3+0];
        temp[1] = output[curb*3+1];
        temp[2] = output[curb*3+2];
    }
	if (yt < (ht-1) and yt > 0 and xt < (wt-1) and xt > 0 and mask[curt] > 127.0f) {
		float gradient[3];
		gradient[0] = 4*target[curt*3+0]-(target[(curt-wt)*3+0]+target[(curt+wt)*3+0]+target[(curt-1)*3+0]+target[(curt+1)*3+0]);
		gradient[1] = 4*target[curt*3+1]-(target[(curt-wt)*3+1]+target[(curt+wt)*3+1]+target[(curt-1)*3+1]+target[(curt+1)*3+1]);
		gradient[2] = 4*target[curt*3+2]-(target[(curt-wt)*3+2]+target[(curt+wt)*3+2]+target[(curt-1)*3+2]+target[(curt+1)*3+2]);
		if (0 < yb and yb < (hb-1) and 0 < xb and xb < (wb-1)) {
			temp[0] = 0.25*(gradient[0]+output[(curb-wb)*3+0]+output[(curb+wb)*3+0]+output[(curb-1)*3+0]+output[(curb+1)*3+0]);
			temp[1] = 0.25*(gradient[1]+output[(curb-wb)*3+1]+output[(curb+wb)*3+1]+output[(curb-1)*3+1]+output[(curb+1)*3+1]);
			temp[2] = 0.25*(gradient[2]+output[(curb-wb)*3+2]+output[(curb+wb)*3+2]+output[(curb-1)*3+2]+output[(curb+1)*3+2]);
		}
	}
    __syncthreads();
    if(0 <= yb and yb < hb and 0 <= xb and xb < wb) {
        output[curb*3+0] = temp[0];
        output[curb*3+1] = temp[1];
        output[curb*3+2] = temp[2];
    }
    __syncthreads();
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	for(int i = 0; i != 20000; ++i)
	SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		background, target, mask, output,
		wb, hb, wt, ht, oy, ox
	);
}
