#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <tuple>
#include <cstdio>
#include <cstdlib>
#include <functional>
#include <algorithm>
#include "SyncedMemory.h"
#include "Timer.h"
#include "counting.h"
using namespace std;

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

template <typename Engine>
tuple<vector<char>, vector<int>, vector<int>> GenerateTestCase(Engine &eng, const int N) {
	poisson_distribution<int> pd(14.0);
	bernoulli_distribution bd(0.1);
	uniform_int_distribution<int> id1(1, 20);
	uniform_int_distribution<int> id2(1, 5);
	uniform_int_distribution<int> id3('a', 'z');
	tuple<vector<char>, vector<int>, vector<int>> ret;
	auto &text = get<0>(ret);
	auto &pos = get<1>(ret);
	auto &head = get<2>(ret);
	auto gen_rand_word_len = [&] () -> int {
		return max(1, min(500, pd(eng) - 5 + (bd(eng) ? id1(eng)*20 : 0)));
	};
	auto gen_rand_space_len = [&] () -> int {
		return id2(eng);
	};
	auto gen_rand_char = [&] () {
		return id3(eng);
	};
	auto AddWord = [&] () {
		head.push_back(text.size());
		int n = gen_rand_word_len();
		for (int i = 0; i < n; ++i) {
			text.push_back(gen_rand_char());
			pos.push_back(i+1);
		}
	};
	auto AddSpace = [&] () {
		int n = gen_rand_space_len();
		for (int i = 0; i < n; ++i) {
			text.push_back('\n');
			pos.push_back(0);
		}
	};

	AddWord();
	while (text.size() < N) {
		AddSpace();
		AddWord();
	}
	return ret;
}

int main(int argc, char **argv)
{
	// Initialize random text
	default_random_engine engine(12345);
	auto text_pos_head = GenerateTestCase(engine, 40000000); // 40 MB data
	vector<char> &text = get<0>(text_pos_head);
	vector<int> &pos = get<1>(text_pos_head);
	vector<int> &head = get<2>(text_pos_head);

	// Prepare buffers
	int n = text.size();
	char *text_gpu;
	hipMalloc(&text_gpu, sizeof(char)*n);
	SyncedMemory<char> text_sync(text.data(), text_gpu, n);
	text_sync.get_cpu_wo(); // touch the cpu data
	MemoryBuffer<int> pos_yours(n), head_yours(n);
	auto pos_yours_sync = pos_yours.CreateSync(n);
	auto head_yours_sync = head_yours.CreateSync(n);

	// Create timers
	Timer timer_count_position;

	// Part I
	timer_count_position.Start();
	int *pos_yours_gpu = pos_yours_sync.get_gpu_wo();
	hipMemset(pos_yours_gpu, 0, sizeof(int)*n);
	CountPosition(text_sync.get_gpu_ro(), pos_yours_gpu, n);
	CHECK;
	timer_count_position.Pause();
	printf_timer(timer_count_position);

	// Part I check
	const int *golden = pos.data();
	const int *yours = pos_yours_sync.get_cpu_ro();
	int n_match1 = mismatch(golden, golden+n, yours).first - golden;
	if (n_match1 != n) {
		puts("Part I WA!");
		copy_n(golden, n, pos_yours_sync.get_cpu_wo());
	}

	// Part II
	int *head_yours_gpu = head_yours_sync.get_gpu_wo();
	hipMemset(head_yours_gpu, 0, sizeof(int)*n);
	int n_head = ExtractHead(pos_yours_sync.get_gpu_ro(), head_yours_gpu, n);
	CHECK;

	// Part II check
	do {
		if (n_head != head.size()) {
			n_head = head.size();
			puts("Part II WA (wrong number of heads)!");
		} else {
			int n_match2 = mismatch(head.begin(), head.end(), head_yours_sync.get_cpu_ro()).first - head.begin();
			if (n_match2 != n_head) {
				puts("Part II WA (wrong heads)!");
			} else {
				break;
			}
		}
		copy_n(head.begin(), n_head, head_yours_sync.get_cpu_wo());
	} while(false);

	// Part III
	// Do whatever your want
	Part3(text_gpu, pos_yours_sync.get_gpu_rw(), head_yours_sync.get_gpu_rw(), n, n_head);
	CHECK;

	hipFree(text_gpu);
	return 0;
}
